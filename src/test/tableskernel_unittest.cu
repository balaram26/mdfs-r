#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <stdint.h>

#include "../gpu/tableskernel.cuh"

#define TILE_SIZE 16
#define DIM 4
#define DIV 3
#define BITS 2

void cpuKernel(KernelParam param) {
	int dim = param.dim - 1;
	int varlen = param.packs[0] + param.packs[1];
	int ncnt = (int)pow(param.div + 1, dim);
	int ntup = (int)pow(param.tileSize, dim);
	int bits = (int)std::ceil(std::log2f((float)(param.div + 1)));

	uint32_t counters[ncnt][2];

	uint64_t* data[dim];
	for (int i = 0; i < dim; i++) {
		data[i] = param.data[i >= dim - param.index ? i + 1 : i];
	}

	// po dyskretyzacjach
	for (int r = 0; r < param.disc; r++) {
		printf("d%d\n", r);
		// po krotkach
		for (int t = 0; t < ntup; t++) {
			for (int cnt = 0; cnt < ncnt; cnt++) {
				counters[cnt][0] = counters[cnt][1] = 0;
			}

			// po decyzjach
			for (int d = 0; d < 2; d++) {
				// po paczkach
				int done = 0;
				for (int p = d ? param.packs[0] : 0;
					p < param.packs[d] + (d ? param.packs[0] : 0);
					p += bits) {
					// po obiektach
					for (int o = 0; done + o < param.objs[d] && o < 64; o++) {
						int pos = 0;
						int shift = 1;

						int tt = t;
						// po współrzędnych
						for (int v = 0; v < dim; v++) {
							int val = 0;
							int var = tt % param.tileSize;
							tt /= param.tileSize;
							// po bitach
							for (int b = 0; b < bits; b++) {
								val |= ((data[v][var * varlen + p + b] >> o) & 1) << b;
							}
							pos += shift * val;
							shift *= param.div + 1;
						}

						counters[pos][d]++;
					}
					done += 64;
				}
			}

			// memcpy? bigendian.
			for (int cnt = 0; cnt < ncnt; cnt++) {
				param.counters[param.index][t * ncnt + cnt] =
					(uint64_t)counters[cnt][0] << 32 | (uint64_t)counters[cnt][1];
				//printf(">>> %x %x %llx\n", (uint64_t)counters[cnt][0], (uint64_t)counters[cnt][1],
				//	param.counters[param.index][t * ncnt + cnt]);
			}
		}

		param.counters[param.index] += ntup * ncnt;
		for (int i = 0; i < dim; i++) {
			data[i] += param.tileSize * varlen;
		}
	}

}

void generateInput(uint64_t* data, int div, int bits, int disc, int vars, int obj0, int obj1) {
	int objs[2] = {obj0, obj1};
	int off[2] = {0, bits * ((obj0 + 63) / 64)};

	int varlen = bits * (((obj0 + 63) / 64) + ((obj1 + 63) / 64));
	int disclen = varlen * vars;

	for (int r = 0; r < disc; r++) {
		for (int v = 0; v < vars; v++) {
			for (int d = 0; d < 2; d++) {
				for (int o = 0; o < objs[d]; o++) {
					int val = rand() % (div + 1);
					for (int b = 0; b < bits; b++) {
						data[r * disclen + v * varlen + off[d] + bits * (o / 64) + b] |=
							(uint64_t)((val >> b) & 1) << o;
					}
				}
			}
		}
	}
}


int main() {
	KernelParam input(TILE_SIZE, DIM, DIV, RM_AVG, BF_SPLIT, true, 0);
	printf("input init... Done.\n");

	input.disc = 3;

	//input.vars = 12345; // WSZYSTKIE zmienne

	// Ile bitów na obiekt?
	int bits = (int)std::ceil(std::log2f((float)(input.div + 1)));
	printf("bits: %d, BITS: %d\n", bits, BITS);

	// Nieistotne:
	// Alokujemy dopełnione do tileSize
	//std::size_t len = (input.vars + input.tileSize - 1) / input.tileSize * input.tileSize;
	//hipMalloc(&input.IG, len * sizeof(float));
	//hipMemset(input.IG, 0, len * sizeof(float));

	// Nieistotne:
	//input.pseudo[0] = 10.0;
	//input.pseudo[1] = 10.0;

	input.objs[0] = 12342;
	input.objs[1] = 33312;

	for (int dec = 0; dec < 2; dec++) {
		input.packs[dec] = ((input.objs[dec] + 63) / 64) * bits;
		printf("input.packs[%d] = %d\n", dec, input.packs[dec]);
	}

	uint64_t vol = 1;
	for (int i = 1; i < input.dim; i++) {
		vol *= input.tileSize * (input.div + 1);
	}
	printf("vol: %d\n", vol);

	printf("hipMalloc...");
	uint64_t varlen = input.packs[0] + input.packs[1];
	printf("varlen: %d\n", varlen);
	uint64_t vars = input.disc * input.tileSize;
	for (int i = 0; i < input.dim; i++) {
		//input.offset[i] = 0;
		hipMalloc(&input.data[i], vars * varlen * sizeof(uint64_t));
		hipMalloc(&input.counters[i], input.disc * vol * sizeof(uint64_t));
	}
	printf(" Done\n");

	printf("generateInput...");
	KernelParam refInput = input;
	for (int i = 0; i < refInput.dim; i++) {
		refInput.data[i] = new uint64_t[vars * varlen];
		memset(refInput.data[i], 0, vars * varlen * sizeof(uint64_t));
		refInput.counters[i] = new uint64_t[input.disc * vol];

		generateInput(refInput.data[i], input.div, bits, refInput.disc, refInput.tileSize, input.objs[0], input.objs[1]);
		hipMemcpy(input.data[i], refInput.data[i], vars * varlen * sizeof(uint64_t), hipMemcpyHostToDevice);
	}
	printf(" Done\n");

	// RUN
	uint64_t* buffer = new uint64_t[input.disc * vol];
	for (int ix = 0; ix < input.dim; ix++) {
		refInput.index = ix;
		input.index = ix;

		cpuKernel(refInput);
		tablesKernelWrapper<TILE_SIZE, DIM - 1, DIV, BITS, 1>(input, 0);

		printf("%d: ", ix);
		hipMemcpy(buffer, input.counters[ix], input.disc * vol * sizeof(uint64_t), hipMemcpyDeviceToHost);
		for (int i = 0; i < input.disc * vol; i++) {
			if (buffer[i] != refInput.counters[ix][i]) {
				printf("FAIL(%d %d) %lx %lx\n", ix, i, buffer[i], refInput.counters[ix][i]);
				exit(1);
			}
		}
		printf("OK\n");
	}

	return 0;
}
